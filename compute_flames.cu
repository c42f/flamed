#include "hip/hip_runtime.h"
#define COMPILE_FOR_GPU
#include "compute_flames.h"

#include <iostream>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cuda_gl_interop.h>
#include <thrust/device_vector.h>

#define ASSERT_CUDA_SUCCESS(expr)                                              \
    do {                                                                       \
        hipError_t err = expr;                                                \
        if(err != hipSuccess)                                                 \
        {                                                                      \
            std::cerr << "Cuda error: \"" << hipGetErrorString(err)           \
                << "\" in calling " #expr "\n";                                \
            abort();                                                           \
        }                                                                      \
    } while(false)

#define ASSERT_KERNEL_SUCCESS(str)                                             \
    do {                                                                       \
        hipError_t err = hipGetLastError();                                  \
        if(err != hipSuccess)                                                 \
        {                                                                      \
            std::cerr << "Cuda error: \"" << hipGetErrorString(err)           \
                << "\" in calling " str "\n";                                  \
            abort();                                                           \
        }                                                                      \
    } while(false)


// TODO: Fix this awful hard coded maximum!
#define MAX_MAPS 20

void initCuda()
{
    ASSERT_CUDA_SUCCESS(cudaGLSetGLDevice(0));
}


__global__ void rngInitKernel(hiprandState_t* generators, int n)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i >= n)
        return;
    hiprand_init(42, i, i, &generators[i]);
}


__global__ void flameGenKernel(IFSPoint* points, hiprandState_t* rngs,
                               int nThreads, long long nPoints,
                               FlameMapping* flameMaps, int nMaps)
{
    long long id = blockIdx.x*blockDim.x + threadIdx.x;
    __shared__ FlameMapping maps[MAX_MAPS];
    if(threadIdx.x < nMaps+1)
        maps[threadIdx.x] = flameMaps[threadIdx.x];
    syncthreads();
    hiprandState_t gen = rngs[id];
    const int discard = 20;
    V2f p(0);
    C3f col(0);
    for(int i = 0; i < discard; ++i)
    {
        float rnd = hiprand_uniform(&gen);
        int mapIdx = int(rnd*nMaps);
        const FlameMapping& m = maps[mapIdx];
        p = m.map(p);
        col = m.colorSpeed*m.col + (1-m.colorSpeed)*col;
    }
    for(long long i = id; i < nPoints; i += nThreads)
    {
        float rnd = hiprand_uniform(&gen);
        int mapIdx = int(rnd*nMaps);
        const FlameMapping& m = maps[mapIdx];
        p = m.map(p);
        col = m.colorSpeed*m.col + (1-m.colorSpeed)*col;
        points[i].pos = maps[nMaps].map(p); // final map is last one
        points[i].col = col;
    }
    rngs[id] = gen;
}


struct GPUFlameEngine::Pimpl
{
    int nThreads;
    thrust::device_vector<FlameMapping> flameMaps;
    thrust::device_vector<hiprandState_t> randState;

    Pimpl(int nThreads)
        : nThreads(nThreads),
        flameMaps(),
        randState(nThreads)
    { }
};


GPUFlameEngine::GPUFlameEngine()
    : m_pimpl(new Pimpl(40000))
{
    const int blockSize = 256;
    rngInitKernel<<<ceildiv(m_pimpl->nThreads,blockSize), blockSize>>>(
        thrust::raw_pointer_cast(&m_pimpl->randState[0]), m_pimpl->nThreads);
    ASSERT_KERNEL_SUCCESS("rngInitKernel");
}


void GPUFlameEngine::generate(PointVBO* points, const FlameMaps& flameMaps)
{
    // Can't get the new API to work for some reason...
//    hipGraphicsResource_t *cudaRes = 0;
//    ASSERT_CUDA_SUCCESS(hipGraphicsGLRegisterBuffer(cudaRes, points->id(),
//                                    hipGraphicsRegisterFlagsWriteDiscard));
//    ASSERT_CUDA_SUCCESS(hipGraphicsMapResources(1, cudaRes));
//    size_t nBytes = 0;
//    ASSERT_CUDA_SUCCESS(hipGraphicsResourceGetMappedPointer((void**)&ifsPoints,
//                                                             &nBytes, *cudaRes));
//    assert(nBytes/sizeof(IFSPoint) == points->size());
//    ASSERT_CUDA_SUCCESS(hipGraphicsUnmapResources(1, cudaRes));
//    ASSERT_CUDA_SUCCESS(hipGraphicsUnregisterResource(*cudaRes));

    ASSERT_CUDA_SUCCESS(cudaGLRegisterBufferObject(points->id()));

    thrust::device_vector<FlameMapping>& flameMaps_d = m_pimpl->flameMaps;
    flameMaps_d = flameMaps.maps;
    flameMaps_d.push_back(flameMaps.finalMap);
    if(flameMaps_d.size() > MAX_MAPS)
        flameMaps_d.resize(MAX_MAPS);

    IFSPoint* ifsPoints = 0;
    ASSERT_CUDA_SUCCESS(cudaGLMapBufferObject((void**)&ifsPoints, points->id()));

    const int blockSize = 256;
    flameGenKernel<<<ceildiv(m_pimpl->nThreads, blockSize), blockSize>>>(
        ifsPoints, thrust::raw_pointer_cast(&m_pimpl->randState[0]),
        m_pimpl->nThreads, points->size(),
        thrust::raw_pointer_cast(&flameMaps_d[0]),
        flameMaps.maps.size()
    );
    ASSERT_KERNEL_SUCCESS("flameGenKernel");

    ASSERT_CUDA_SUCCESS(cudaGLUnmapBufferObject(points->id()));
    ASSERT_CUDA_SUCCESS(cudaGLUnregisterBufferObject(points->id()));
}

